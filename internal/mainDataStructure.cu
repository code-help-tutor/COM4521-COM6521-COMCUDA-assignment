#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cctype>
#include <cstring>
#include <cstdint>
#include <ctime>
#include <random>

#include "config.h"
#include "common.h"
#include "cpu.h"
#include "openmp.h"
#include "cuda.cuh"
#include "main.h"

namespace {
/**
 * Structure containing the options provided by runtime arguments
 */
struct DSConfig {
    /**
      * Path to input file
      */
    char *input_file = nullptr;
    /**
      * Path to output file
      */
    char *output_file = nullptr;
    /**
      * Random seed
      * If neither input_file nor random_seed are used, time will be used as seed
      */
    unsigned int random_seed = 0;
    /**
      * Random length
      * If generating random data, this is how much data will be generated
      */
    unsigned int random_length = 1000000;
    /**
      * Program will operate in benchmark mode
      * This repeats the algorithm multiple times and returns an average time
      */
    bool benchmark;
}; typedef struct DSConfig DSConfig;
/**
 * Parse the runtime args into config
 * @param argc argc from main()
 * @param argv argv from main()]
 * @param config Pointer to config structure for return value
 */
void parse_args(int argc, char** argv, DSConfig* config) {
    // Clear config struct
    *config = {};
    // Iterate over remaining args    
    int i = 3;
    char* t_arg = 0;
    for (; i < argc; i++) {
        // Make a lowercase copy of the argument
        const size_t arg_len = strlen(argv[i]) + 1;  // Add 1 for null terminating character
        if (t_arg)
            free(t_arg);
        t_arg = (char*)malloc(arg_len);
        int j = 0;
        for (; argv[i][j]; ++j) {
            t_arg[j] = tolower(argv[i][j]);
        }
        t_arg[j] = '\0';
        // Decide which arg it is
        // Benchmark
        if (!strcmp("--bench", t_arg) || !strcmp("--benchmark", t_arg) || !strcmp("-b", t_arg)) {
            config->benchmark = 1;
            continue;
        }
        // Input file
        if (!strcmp(t_arg + arg_len - 5, ".csv")) {
            if (config->input_file  || (config->random_seed && config->random_length)) {
                if (config->output_file) {
                    fprintf(stderr, "Multiple inputs/outputs were provided, this is not supported!\n");
                    print_help(argv[0]);
                } else {
                    // Allocate memory and copy
                    config->output_file = (char*)malloc(arg_len);
                    memcpy(config->output_file, argv[i], arg_len);
                    continue;
                }
            } else {
                // Allocate memory and copy
                config->input_file = (char*)malloc(arg_len);
                memcpy(config->input_file, argv[i], arg_len);
                continue;
            }
        }
        // Random seed + length
        if (i + 1 < argc) {
            // Random seed
            char* end = nullptr;
            const unsigned int t_arg_uint = (unsigned int)strtoul(argv[i], &end, 10);
            // Test that it converts back to the same string as a form of validation
            const int n = snprintf(NULL, 0, "%u", t_arg_uint);
            if (n > 0) {
                char* buf = (char*)malloc(n + 1);
                int c = snprintf(buf, n + 1, "%u", t_arg_uint);
                if (!strcmp(buf, argv[i])) {
                    if (config->random_seed) {
                        fprintf(stderr, "Multiple random seeds were provided, this is not supported!\n");
                        print_help(argv[0]);
                    }
                    config->random_seed = t_arg_uint;
                }
                free(buf);
            }
            if (config->random_seed) {  // Length
                ++i;
                char* end = nullptr;
                const unsigned int t_arg_uint = (unsigned int)strtoul(argv[i], &end, 10);
                // Test that it converts back to the same string as a form of validation
                const int n = snprintf(NULL, 0, "%u", t_arg_uint);
                if (n > 0) {
                    char* buf = (char*)malloc(n + 1);
                    int c = snprintf(buf, n + 1, "%u", t_arg_uint);
                    if (!strcmp(buf, argv[i])) {
                        free(buf);
                        config->random_length = t_arg_uint;
                    } else {
                        free(buf);
                    }
                }
            }
        } else {
            fprintf(stderr, "Unexpected standard deviation argument: %s\n", argv[i]);
            print_help(argv[0]);
        }
    }
    if (config->input_file && config->random_seed) {
        fprintf(stderr, "Both input file and random seed were specified\n");
        print_help(argv[0]);
    } else if (!config->input_file && !config->random_seed) {
        fprintf(stderr, "Neither input file nor random seed/length were specified\n");
        print_help(argv[0]);
    }
    if (t_arg)
        free(t_arg);
}
}

void runDataStructure(int argc, char** argv, const Implementation implementation) {
    DSConfig config;
    parse_args(argc, argv, &config);

    // Inputs
    unsigned int *input_keys = nullptr;
    size_t input_keys_elements = 0;

    // Load/Generate input
    if (config.input_file) {
        // Load CSV
        printf("Using input file: %s%s%s\n", CONSOLE_YELLOW, config.input_file, CONSOLE_RESET);
        loadCSV(config.input_file, reinterpret_cast<void**>(&input_keys), &input_keys_elements, "%u");
        printf("Input has length: %s%u%s\n", CONSOLE_YELLOW, static_cast<unsigned int>(input_keys_elements), CONSOLE_RESET);
    } else {
        // Random init
        if (!config.random_seed) {
            config.random_seed = static_cast<unsigned int>(time(nullptr));
        }
        printf("Using random seed: %s%u%s\n", CONSOLE_YELLOW, config.random_seed, CONSOLE_RESET);
        printf("Generating input of length: %s%u%s\n", CONSOLE_YELLOW, config.random_length, CONSOLE_RESET);
        // Generate a random population
        input_keys_elements = config.random_length;
        input_keys = static_cast<unsigned int*>(malloc(input_keys_elements * sizeof(unsigned int)));
        std::mt19937 rng(config.random_seed);
        std::normal_distribution<float> dist(0.0, 10.0);
        unsigned int num = 0;
        for (unsigned int i = 0; i < input_keys_elements;) {
            const unsigned int count = static_cast<unsigned int>(abs(floor(dist(rng))));
            for (unsigned int j = 0; j < count && i < input_keys_elements; ++j) {
                input_keys[i++] = num;
            }
            ++num;            
        }
    }

    // Create result for validation
    const size_t boundaries_elements = input_keys[input_keys_elements - 1] + 2;
    unsigned int* validation_boundaries = static_cast<unsigned int*>(malloc(boundaries_elements * sizeof(unsigned int)));
    cpu_datastructure(input_keys, input_keys_elements, validation_boundaries, boundaries_elements);

    // Run student implementation
    float timing_log;
    unsigned int* result_boundaries = static_cast<unsigned int*>(malloc(boundaries_elements * sizeof(unsigned int)));
    const int TOTAL_RUNS = config.benchmark ? BENCHMARK_RUNS : 1;
    {
        //Init for run  
        hipEvent_t startT, stopT;
        CUDA_CALL(hipEventCreate(&startT));
        CUDA_CALL(hipEventCreate(&stopT));
        // Run 1 or many times
        timing_log = 0.0f;
        for (int runs = 0; runs < TOTAL_RUNS; ++runs) {
            if (TOTAL_RUNS > 1)
                printf("\r%d/%d", runs + 1, TOTAL_RUNS);
            // Run Adaptive Histogram algorithm
            CUDA_CALL(hipEventRecord(startT));
            CUDA_CALL(hipEventSynchronize(startT));
            switch (implementation) {
            case CPU:
                cpu_datastructure(input_keys, input_keys_elements, result_boundaries, boundaries_elements);
                break;
            case OPENMP:
                openmp_datastructure(input_keys, input_keys_elements, result_boundaries, boundaries_elements);
                break;
            case CUDA:
                cuda_datastructure(input_keys, input_keys_elements, result_boundaries, boundaries_elements);
                break;
            }
            CUDA_CALL(hipEventRecord(stopT));
            CUDA_CALL(hipEventSynchronize(stopT));
            // Sum timing info
            float milliseconds = 0;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, startT, stopT));
            timing_log += milliseconds;
        }
        if (TOTAL_RUNS > 1)
            printf("\n");
        // Convert timing info to average
        timing_log /= TOTAL_RUNS;

        // Cleanup timing
        hipEventDestroy(startT);
        hipEventDestroy(stopT);
    }

    // Validate and report
    {
        unsigned int errors = 0;
        for (unsigned int i = 0; i < boundaries_elements; ++i) {
            if (validation_boundaries[i] != result_boundaries[i]) {
                ++errors;
            }
        }
        printf("Date Structure Result: %s" CONSOLE_RESET "\n", errors ? CONSOLE_RED "Fail" : CONSOLE_GREEN "Pass");
        if (errors) {
            printf("\t%u/%u elements wrong!" CONSOLE_RESET "\n", errors, static_cast<unsigned int>(boundaries_elements));
            printf("\t(Consider comparing output csvs)\n");
        }
    }

    // Export output
    if (config.output_file) {
        saveCSV(config.output_file, result_boundaries, boundaries_elements);
    }
    
    // Report timing information    
    printf("%s average execution timing from %d runs\n", implementation_to_string(implementation), TOTAL_RUNS);
    if (implementation == CUDA) {
        int device_id = 0;
        CUDA_CALL(hipGetDevice(&device_id));
        hipDeviceProp_t props;
        memset(&props, 0, sizeof(hipDeviceProp_t));
        CUDA_CALL(hipGetDeviceProperties(&props, device_id));
        printf("Using GPU: %s\n", props.name);
    }
#ifdef _DEBUG
    printf(CONSOLE_YELLOW "Code built as DEBUG, timing results are invalid!\n" CONSOLE_RESET);
#endif
    printf("Time: %.3fms\n", timing_log);

    // Cleanup
    free(input_keys);
    free(validation_boundaries);
    free(result_boundaries);
    if (config.output_file)
        free(config.output_file);
    if (config.input_file)
        free(config.input_file);
}

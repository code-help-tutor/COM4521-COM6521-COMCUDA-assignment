#include <cstdlib>
#include <cctype>
#include <cstring>
#include <cstdint>
#include <ctime>
#include <random>

#include "config.h"
#include "common.h"
#include "cpu.h"
#include "openmp.h"
#include "cuda.cuh"
#include "main.h"

namespace {
/**
 * Structure containing the options provided by runtime arguments
 */
struct SDConfig {
    /**
      * Path to input file
      */
    char *input_file = nullptr;
    /**
      * Random seed
      * If neither input_file nor random_seed are used, time will be used as seed
      */
    unsigned int random_seed = 0;
    /**
      * Random length
      * If generating random data, this is how much data will be generated
      */
    unsigned int random_length = 1000000;
    /**
      * Program will operate in benchmark mode
      * This repeats the algorithm multiple times and returns an average time
      */
    bool benchmark;
}; typedef struct SDConfig SDConfig;
/**
 * Parse the runtime args into config
 * @param argc argc from main()
 * @param argv argv from main()]
 * @param config Pointer to config structure for return value
 */
void parse_args(int argc, char** argv, SDConfig* config) {
    // Clear config struct
    *config = {};
    // Iterate over remaining args    
    int i = 3;
    char* t_arg = 0;
    for (; i < argc; i++) {
        // Make a lowercase copy of the argument
        const size_t arg_len = strlen(argv[i]) + 1;  // Add 1 for null terminating character
        if (t_arg)
            free(t_arg);
        t_arg = (char*)malloc(arg_len);
        int j = 0;
        for (; argv[i][j]; ++j) {
            t_arg[j] = tolower(argv[i][j]);
        }
        t_arg[j] = '\0';
        // Decide which arg it is
        // Benchmark
        if (!strcmp("--bench", t_arg) || !strcmp("--benchmark", t_arg) || !strcmp("-b", t_arg)) {
            config->benchmark = 1;
            continue;
        }
        // Input file
        if (!strcmp(t_arg + arg_len - 5, ".csv")) {
            if (config->input_file) {
                fprintf(stderr, "Multiple inputs were provided, this is not supported!\n");
                print_help(argv[0]);
            } else {
                // Allocate memory and copy
                config->input_file = (char*)malloc(arg_len);
                memcpy(config->input_file, argv[i], arg_len);
                continue;
            }
        }
        // Random seed + length
        if (i + 1 < argc) {
            // Random seed
            char* end = nullptr;
            const unsigned int t_arg_uint = (unsigned int)strtoul(argv[i], &end, 10);
            // Test that it converts back to the same string as a form of validation
            const int n = snprintf(NULL, 0, "%u", t_arg_uint);
            if (n > 0) {
                char* buf = (char*)malloc(n + 1);
                int c = snprintf(buf, n + 1, "%u", t_arg_uint);
                if (!strcmp(buf, argv[i])) {
                    if (config->random_seed) {
                        fprintf(stderr, "Multiple random seeds were provided, this is not supported!\n");
                        print_help(argv[0]);
                    }
                    config->random_seed = t_arg_uint;
                }
                free(buf);
            }
            if (config->random_seed) {  // Length
                ++i;
                char* end = nullptr;
                const unsigned int t_arg_uint = (unsigned int)strtoul(argv[i], &end, 10);
                // Test that it converts back to the same string as a form of validation
                const int n = snprintf(NULL, 0, "%u", t_arg_uint);
                if (n > 0) {
                    char* buf = (char*)malloc(n + 1);
                    int c = snprintf(buf, n + 1, "%u", t_arg_uint);
                    if (!strcmp(buf, argv[i])) {
                        free(buf);
                        config->random_length = t_arg_uint;
                    } else {
                        free(buf);
                    }
                }
            }
        } else {
            fprintf(stderr, "Unexpected standard deviation argument: %s\n", argv[i]);
            print_help(argv[0]);
        }
    }
    if (config->input_file && config->random_seed) {
        fprintf(stderr, "Both input file and random seed were specified\n");
        print_help(argv[0]);
    } else if (!config->input_file && !config->random_seed) {
        fprintf(stderr, "Neither input file nor random seed/length were specified\n");
        print_help(argv[0]);
    }
    if (t_arg)
        free(t_arg);
}
}

void runStandardDeviation(int argc, char** argv, const Implementation implementation) {
    SDConfig config;
    parse_args(argc, argv, &config);

    // Inputs
    float *input_buffer = nullptr;
    size_t input_buffer_elements = 0;

    // Load/Generate input
    if (config.input_file) {
        // Load CSV
        printf("Using input file: %s%s%s\n", CONSOLE_YELLOW, config.input_file, CONSOLE_RESET);
        loadCSV(config.input_file, reinterpret_cast<void**>(&input_buffer), &input_buffer_elements, "%f");
        printf("Input has length: %s%u%s\n", CONSOLE_YELLOW, static_cast<unsigned int>(input_buffer_elements), CONSOLE_RESET);
    } else {
        // Random init
        if (!config.random_seed) {
            config.random_seed = static_cast<unsigned int>(time(nullptr));
        }
        printf("Using random seed: %s%u%s\n", CONSOLE_YELLOW, config.random_seed, CONSOLE_RESET);
        printf("Generating input of length: %s%u%s\n", CONSOLE_YELLOW, config.random_length, CONSOLE_RESET);
        // Generate a random population
        input_buffer_elements = config.random_length;
        input_buffer = static_cast<float*>(malloc(input_buffer_elements * sizeof(float)));
        std::mt19937 rng(config.random_seed);
        std::uniform_real_distribution<float> dist(-1000.0f, 1000.0f);
        for (unsigned int i = 0; i < input_buffer_elements; ++i) {
            input_buffer[i] = dist(rng);
        }
    }

    // Create result for validation
    const float standard_deviation_validation = cpu_standarddeviation(input_buffer, input_buffer_elements);

    // Run student implementation
    float timing_log;
    float standard_deviation_result = -1;
    const int TOTAL_RUNS = config.benchmark ? BENCHMARK_RUNS : 1;
    {
        //Init for run  
        hipEvent_t startT, stopT;
        CUDA_CALL(hipEventCreate(&startT));
        CUDA_CALL(hipEventCreate(&stopT));
        // Run 1 or many times
        timing_log = 0.0f;
        for (int runs = 0; runs < TOTAL_RUNS; ++runs) {
            if (TOTAL_RUNS > 1)
                printf("\r%d/%d", runs + 1, TOTAL_RUNS);
            // Run Adaptive Histogram algorithm
            CUDA_CALL(hipEventRecord(startT));
            CUDA_CALL(hipEventSynchronize(startT));
            switch (implementation) {
            case CPU:
                standard_deviation_result = cpu_standarddeviation(static_cast<float*>(input_buffer), input_buffer_elements);
                break;
            case OPENMP:
                standard_deviation_result = openmp_standarddeviation(static_cast<float*>(input_buffer), input_buffer_elements);
                break;
            case CUDA:
                standard_deviation_result = cuda_standarddeviation(static_cast<float*>(input_buffer), input_buffer_elements);
                break;
            }
            CUDA_CALL(hipEventRecord(stopT));
            CUDA_CALL(hipEventSynchronize(stopT));
            // Sum timing info
            float milliseconds = 0;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, startT, stopT));
            timing_log += milliseconds;
        }
        if (TOTAL_RUNS > 1)
            printf("\n");
        // Convert timing info to average
        timing_log /= TOTAL_RUNS;

        // Cleanup timing
        hipEventDestroy(startT);
        hipEventDestroy(stopT);
    }

    // Validate and report
    {
        const bool FAIL = !equalsEpsilon(standard_deviation_validation, standard_deviation_result, 0.1f);
        printf("Standard Deviation Result: %s (epsilon 0.1)" CONSOLE_RESET "\n", FAIL ? CONSOLE_RED "Fail" : CONSOLE_GREEN "Pass");
            printf("\tCPU: " CONSOLE_YELLOW "%.2f" CONSOLE_RESET "\n", standard_deviation_validation);
            printf("\t%s: %s%.2f" CONSOLE_RESET "\n", implementation_to_string(implementation), FAIL ? CONSOLE_RED : CONSOLE_GREEN, standard_deviation_result);
    }

    // Export output
    // Nothing to export, results are printed to stdout
    
    // Report timing information    
    printf("%s average execution timing from %d runs\n", implementation_to_string(implementation), TOTAL_RUNS);
    if (implementation == CUDA) {
        int device_id = 0;
        CUDA_CALL(hipGetDevice(&device_id));
        hipDeviceProp_t props;
        memset(&props, 0, sizeof(hipDeviceProp_t));
        CUDA_CALL(hipGetDeviceProperties(&props, device_id));
        printf("Using GPU: %s\n", props.name);
    }
#ifdef _DEBUG
    printf(CONSOLE_YELLOW "Code built as DEBUG, timing results are invalid!\n" CONSOLE_RESET);
#endif
    printf("Time: %.3fms\n", timing_log);

    // Cleanup
    if (input_buffer)
        free(input_buffer);
    if (config.input_file)
        free(config.input_file);
}
